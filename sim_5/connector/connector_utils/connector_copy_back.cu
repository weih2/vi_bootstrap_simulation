void connector::connector_copy_back(){
  for(int k = 0; k < N_CENTERS; k++){
    hipMemcpy(credible_sets_lengths[k], device_credible_sets_lengths + k * N_EXPERIMENTS,
      sizeof(double) * N_EXPERIMENTS, hipMemcpyDeviceToHost);
    hipMemcpy(credible_sets_covered[k], device_credible_sets_covered + k * N_EXPERIMENTS,
      sizeof(int) * N_EXPERIMENTS, hipMemcpyDeviceToHost);
  }
  hipMemcpy(fi_inv, dev_fi_inv,
    sizeof(double) * N_CENTERS * N_CENTERS * N_EXPERIMENTS, hipMemcpyDeviceToHost);

  hipMemcpy(first_center, dev_first_center, sizeof(double) * N_EXPERIMENTS, hipMemcpyDeviceToHost);
}
