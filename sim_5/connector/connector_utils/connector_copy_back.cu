void connector::connector_copy_back(){
  for(int k = 0; k < N_CENTERS; k++){
    hipMemcpy(credible_sets_lengths[k], device_credible_sets_lengths[k],
      sizeof(double) * N_EXPERIMENTS, hipMemcpyDeviceToHost);
    hipMemcpy(credible_sets_covered[k], device_credible_sets_covered[k],
      sizeof(int) * N_EXPERIMENTS, hipMemcpyDeviceToHost);
  }
}
