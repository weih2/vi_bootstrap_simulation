connector::connector(){
  for(int k = 0; k < N_CENTERS; k++){
    hipMalloc((void**)& (device_credible_sets_lengths[k]), N_EXPERIMENTS * sizeof(double));
    hipMalloc((void**)& (device_credible_sets_covered[k]), N_EXPERIMENTS * sizeof(int));
  }
}

connector::~connector(){
  for(int k = 0; k < N_CENTERS; k++){
    hipFree(device_credible_sets_lengths[k]);
    hipFree(device_credible_sets_covered[k]);
  }
}
