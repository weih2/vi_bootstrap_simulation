connector::connector(){
  for(int k = 0; k < N_CENTERS; k++){
    hipMalloc((void**)& device_credible_sets_lengths, N_CENTERS * N_EXPERIMENTS * sizeof(double));
    hipMalloc((void**)& device_credible_sets_covered, N_CENTERS * N_EXPERIMENTS * sizeof(int));
  }
}

connector::~connector(){
    hipFree(device_credible_sets_lengths);
    hipFree(device_credible_sets_covered);
}
