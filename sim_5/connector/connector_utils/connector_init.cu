connector::connector(){
  for(int k = 0; k < N_CENTERS; k++){
    hipMalloc((void**)& device_credible_sets_lengths, N_CENTERS * N_EXPERIMENTS * sizeof(double));
    hipMalloc((void**)& device_credible_sets_covered, N_CENTERS * N_EXPERIMENTS * sizeof(int));
  }
  // hipMalloc((void**)& dev_fi_inv, N_CENTERS * N_CENTERS * N_EXPERIMENTS * sizeof(double));
}

connector::~connector(){
    hipFree(device_credible_sets_lengths);
    hipFree(device_credible_sets_covered);
    // hipFree(dev_fi_inv);
}
