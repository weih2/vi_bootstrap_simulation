connector::connector(){
  for(int k = 0; k < N_CENTERS; k++){
    hipMalloc((void**)& device_credible_sets_lengths, N_CENTERS * N_EXPERIMENTS * sizeof(double));
    hipMalloc((void**)& device_credible_sets_covered, N_CENTERS * N_EXPERIMENTS * sizeof(int));
  }
  hipMalloc((void**)& dev_fi_inv, N_CENTERS * N_CENTERS * N_EXPERIMENTS * sizeof(double));
  for(int n = 0; n < N_EXPERIMENTS; n++)
    dev_fi_inv_[n] = dev_fi_inv + N_CENTERS * N_CENTERS * n;
}

connector::~connector(){
    printf("%f\n", dev_fi_inv_[0][0]);
    hipFree(device_credible_sets_lengths);
    hipFree(device_credible_sets_covered);
    hipFree(dev_fi_inv_);
}
