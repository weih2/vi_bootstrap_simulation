// take inversion of sample covariance matrices
// which is approximation of Fisher information
void connector::invert_fi_back(){
  // initialize a handle
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // useless arrays..
  int *p_arr, *info_arr;
  // temporarily store fisher info estimate
  double *device_fi[N_EXPERIMENTS], *device_fi_;

  hipMalloc((void**)&p_arr, N_CENTERS * N_EXPERIMENTS * sizeof(int));
  hipMalloc((void**)&info_arr, N_CENTERS * N_EXPERIMENTS * sizeof(int));
  // exist in global memory
  hipMalloc((void**)&device_fi_,
    N_CENTERS * N_CENTERS * N_EXPERIMENTS * sizeof(double));

  for(int n = 0; n < N_EXPERIMENTS; n++)
    device_fi[n] = device_fi_ + N_CENTERS * N_CENTERS * n;


  // LU decompositions
  hipblasDgetrfBatched(handle, N_CENTERS, dev_fi_inv_,
    N_CENTERS, p_arr, info_arr, N_EXPERIMENTS);
  // inversion
  hipblasDgetriBatched(handle, N_CENTERS, dev_fi_inv_, N_CENTERS, p_arr,
    device_fi, N_CENTERS, info_arr, N_EXPERIMENTS);

  // copy back to host
  hipMemcpy(fi_est, device_fi_,
    N_CENTERS * N_CENTERS * N_EXPERIMENTS * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(p_arr);
  hipFree(info_arr);
  hipFree(device_fi_);
  hipblasDestroy(handle);
}
