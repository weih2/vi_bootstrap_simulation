void connector::run_mcmc(double delta){
  connect_to_execution<<<32,32>>>(delta, device_credible_sets_lengths, device_credible_sets_covered);
  hipDeviceSynchronize();
  connector_copy_back();
  hipDeviceSynchronize();
}
