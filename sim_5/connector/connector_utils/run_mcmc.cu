void connector::run_mcmc(double delta){
  connect_to_execution<<<32,32>>>(delta, device_credible_sets_lengths, device_credible_sets_covered,
    dev_fi_inv_);
  hipDeviceSynchronize();
  connector_copy_back();
  hipDeviceSynchronize();
  invert_fi_back();
  hipDeviceSynchronize();
}
