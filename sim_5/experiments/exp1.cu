#include "hip/hip_runtime.h"
#include "../include.cuh"

int main(){
  double test_seq;

  connector executor = connector();

  for(int m = 0; m < 51; m++){
    test_seq = m>0? (test_seq * exp(0.1)):exp(-3);

    executor.run_mcmc(test_seq);
    executor.gen_stats();
    executor.print_stats();
  }
}
