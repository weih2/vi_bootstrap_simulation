#include "hip/hip_runtime.h"
__global__ void connect_to_execution
(double delta, double *dev_credible_sets_lengths[], int *dev_credible_sets_covered[]){
  int t_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(t_id >= N_EXPERIMENTS) return;

  if(t_id == 1) printf("\n");

  device_mcmc_implementor thread_implementation(delta, t_id);
  thread_implementation.gen_obs();
  thread_implementation.gen_mcmc_samples();
  if(t_id == 1) printf("\n");
  thread_implementation.construct_mcmc_credible_sets();

  if(t_id == 1) {
    printf("\n");
    thread_implementation.print_sample(N_BURN_IN);
  }

  for(int k = 0; k < N_CENTERS; k++){
    dev_credible_sets_covered[k][t_id] = thread_implementation.covered[k];
    dev_credible_sets_lengths[k][t_id] =
    thread_implementation.mcmc_credible_sets[k][1] - thread_implementation.mcmc_credible_sets[k][0];
  }
}
