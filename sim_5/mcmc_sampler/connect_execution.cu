#include "hip/hip_runtime.h"
__global__ void connect_to_execution
(double delta, double *dev_credible_sets_lengths[], int *dev_credible_sets_covered[]){
  int t_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(t_id >= N_EXPERIMENTS) return;

  device_mcmc_implementor thread_implementation(delta, t_id);
  thread_implementation.gen_obs();
  thread_implementation.gen_mcmc_samples();
  thread_implementation.construct_mcmc_credible_sets();

  for(int k = 0; k < N_CENTERS; k++){
    printf("%d\n", t_id);
    dev_credible_sets_covered[k][t_id] = thread_implementation.covered[k];
    dev_credible_sets_lengths[k][t_id] =
    thread_implementation.mcmc_credible_sets[k][1] - thread_implementation.mcmc_credible_sets[k][0];
  }
}
