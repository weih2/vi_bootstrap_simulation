#include "hip/hip_runtime.h"
__global__ void connect_to_execution
(double delta, double *dev_credible_sets_lengths, int *dev_credible_sets_covered){
  int t_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(t_id >= N_EXPERIMENTS) return;

  device_mcmc_implementor thread_implementation(delta, t_id);
  thread_implementation.gen_obs();
  thread_implementation.gen_mcmc_samples();
  thread_implementation.construct_mcmc_credible_sets();

  for(int k = 0; k < N_CENTERS; k++){
    dev_credible_sets_covered[k * N_EXPERIMENTS + t_id] = thread_implementation.covered[k];
    dev_credible_sets_lengths[k * N_EXPERIMENTS + t_id] =
    thread_implementation.mcmc_credible_sets[k][1] - thread_implementation.mcmc_credible_sets[k][0];
  }

  if(t_id == 0){
    printf("%f %f", thread_implementation.mcmc_credible_sets[0][0], thread_implementation.mcmc_credible_sets[0][1]);
  }
}
