__device__ device_mcmc_implementor::fi_inv_estimate(){
  double fi_inv_estimation[N_CENTERS * N_CENTERS]

  for(int k1 = 0; k1 < N_CENTERS; k1++){
    for(int k2 = 0; k2 <= k1; k2++){
      fi_inv_estimation[k1 * N_CENTERS + k2] = cal_covariance(mu_samples[k1], mu_samples[k2],
        N_MCMC_SAMPLES);
      fi_inv_estimation[k2 * N_CENTERS + k1] = fi_inv_estimation[k1 * N_CENTERS + k2];
    }
  }

  hipblasHandle_t cnpHandle;
  hipblasCreate(&cnpHandle);
  int pivot_arr[N_CENTERS];
  int info_cblas;

  // fxxking hard to use
  hipblasDgetrfBatched(cnpHandle, N_CENTERS, fi_inv_estimation,
     N_CENTERS, pivot_arr, &info_cblas, 1);

  hipblasDgetriBatched(cnpHandle, N_CENTERS, fi_inv_estimation, N_CENTERS, pivot_arr,
    fi_estimation, N_CENTERS, &info_cblas, 1);
}
