#include "hip/hip_runtime.h"
__device__ void device_mcmc_implementor::gen_mcmc_samples(){
  hiprandState state;
  hiprand_init(thread_id, 2, 0, &state);

  double mu_sample0[N_CENTERS];
  double cat_prob[N_CENTERS];
  double cat_prob_normalizer;

  double cat_mu_sum[N_CENTERS];
  double cat_mu_count[N_CENTERS];

  double ru;
  double rn;

  double sample_var;
  int sample_count = 0;

  //////// draw a random initial value
  for(int k = 0; k < N_CENTERS; k++)
    mu_sample0[k] = 0;

  //////// burn-in period
  for(int step = 0; step < N_BURN_IN + N_MCMC_SAMPLES * N_INTER; step++){
    for(int k = 0; k < N_CENTERS; k++){
      cat_mu_sum[k] = 0;
      cat_mu_count[k] = 0;
    }
    // sample categorical categorical probabilities
    for(int i = 0; i < N_OBS; i++){
      cat_prob_normalizer = 0;
      for(int k = 0; k < N_CENTERS; k++){
        cat_prob_normalizer += ( cat_prob[k] = exp(- (obs[i] - mu_sample0[k]))/2. );
      }
      ru = hiprand_uniform_double(&state);
      ru /= cat_prob_normalizer;
      for(int k = 0; k < N_CENTERS; k++){
        if(ru < cat_prob[k]){
          cat_mu_count[k]++;
          cat_mu_sum[k] += obs[i];
          if(thread_id == 0) printf("%d \n", cat_mu_count[2]);
          break;
        }
        ru -= cat_prob[k];
      }
    }


    // sample mu
    for(int k = 0; k < N_CENTERS; k++){
      sample_var = PRIOR_SIGMA2/(1 + PRIOR_SIGMA2 * cat_mu_count[k]);
      rn = hiprand_normal_double(&state);
      mu_sample0[k] = rn * sqrt(sample_var) + sample_var * cat_mu_sum[k];
    }
    thrust::sort(thrust::device, mu_sample0, mu_sample0 + N_CENTERS);
    // take sample if
    if(step >= N_BURN_IN){
      if(step % N_INTER == 0){
        for(int k = 0; k < N_CENTERS; k++){
          mu_samples[k][sample_count] = mu_sample0[k];
        }
        sample_count ++;
      }
    }
  }
}
