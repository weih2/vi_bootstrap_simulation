#include "hip/hip_runtime.h"
__device__ void device_mcmc_implementor::gen_obs(){
  int cat;
  hiprandState state;
  hiprand_init(thread_id, 1, 0, &state);

  for(int i = 0; i < N_OBS; i++){
    cat = floor(hiprand_uniform_double(&state) * K);
    obs[i] = mu[cat] + hiprand_normal_double(&state);
  }
}
