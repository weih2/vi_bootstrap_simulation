
#include <hip/hip_runtime.h>
__device__ __host__ inline double cal_mean(const double *observations, int n_observations){
  double mean = 0;
  for(int o = 0; o < n_observations; o++){
    mean += observations[o];
  }
  mean /= double(n_observations);
  return mean;
}

__device__ __host__ inline double cal_variance(const double *observations, int n_observations){
  double mean = cal_mean(observations, n_observations);

  double variance = 0;
  for(int o = 0; o < n_observations; o++){
    variance += (observations[o] - mean) * (observations[o] - mean);
  }
  variance /= double(n_observations - 1);
  return variance;
}

__device__ __host__ inline double cal_covariance(const double *observations1, const double *observations2,
  int n_observations){
  double mean_1 = cal_mean(observations1, n_observations);
  double mean_2 = cal_mean(observations2, n_observations);

  double covariance = 0;
  for(int o = 0; o < n_observations; o++){
    covariance += (observations1[o] - mean_1) * (observations2[o] - mean_2);
  }
  covariance /= double(n_observations - 1);
  return covariance;
}
