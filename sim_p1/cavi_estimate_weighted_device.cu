void cavi_implementation::device_init_cavi_weighted(){
  // copy global setting to device
  hipMalloc((void**)&(device_store.device_g_vars.device_K), sizeof(int));
  hipMemcpy(device_store.device_g_vars.device_K, &data.g_vars.K, sizeof(int), hipMemcpyHostToDevice);
  hipMalloc((void**)&(device_store.device_g_vars.device_sigma_2), sizeof(double));
  hipMemcpy(device_store.device_g_vars.device_sigma_2, &data.g_vars.sigma_2, sizeof(double),
    hipMemcpyHostToDevice);
  hipMalloc((void**)&(device_store.device_g_vars.device_n_samples), sizeof(int));
  hipMemcpy(device_store.device_g_vars.device_n_samples, &data.g_vars.n_samples, sizeof(int),
    hipMemcpyHostToDevice);

  hipMalloc((void**)&device_store.device_n_bootstrap_samples, sizeof(int));
  hipMemcpy(device_store.device_n_bootstrap_samples, &n_bootstrap_samples,
    sizeof(int), hipMemcpyHostToDevice);
  // allocate memory for weights using global memory temporarily
  hipMalloc((void**)&device_store.device_weights,
    n_bootstrap_samples * data.g_vars.n_samples * sizeof(double));
  // allocate memory for device data
  hipMalloc((void**)&device_store.device_x, data.g_vars.n_samples * sizeof(double));
  // allocate memory for device latent variables
  hipMalloc((void**)&(device_store.device_est.device_m),
    n_bootstrap_samples * data.g_vars.K * sizeof(double));
  hipMalloc((void**)&(device_store.device_est.device_s2),
    n_bootstrap_samples * data.g_vars.K * sizeof(double));
  hipMalloc((void**)&(device_store.device_est.device_phi),
    n_bootstrap_samples * data.g_vars.n_samples * data.g_vars.K * sizeof(double));
  // allocate another block of memory for rearranged latent variables
  hipMalloc((void**)&(device_store.device_est.device_m_transpose),
    n_bootstrap_samples * data.g_vars.K * sizeof(double));
  // allocate for device elbo
  hipMalloc((void**)(&device_store.device_elbo), n_bootstrap_samples * sizeof(double));

  // cavi settings
  hipMalloc((void**)(&device_store.device_epsilon), sizeof(double));
  hipMemcpy(device_store.device_epsilon, &epsilon, sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)(&device_store.device_max_n_iter), sizeof(int));
  hipMemcpy(device_store.device_max_n_iter, &max_n_iter, sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)(&device_store.device_exp_id), sizeof(int));
}

/*
The storage is in the spirit of
[n_bootstrap][n_sample][n_variable]
for all variables
except for device_store.device_est.device_m_transpose
*/

void cavi_implementation::cavi_weighted_copy_back(){
  // copy the latent variables back to ram
  hipMemcpy(host_m_transpose, device_store.device_est.device_m_transpose,
     n_bootstrap_samples * data.g_vars.K * sizeof(double),
     hipMemcpyDeviceToHost);
}

void cavi_implementation::device_cavi_bootstrap_update(){
  // yeah only one line
  hipMemcpy(device_store.device_exp_id, &n_experiments, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_store.device_x, data.x, data.g_vars.n_samples * sizeof(double) , hipMemcpyHostToDevice);
  std::cout << "running..." << std::endl;
  device_cavi_bootstrap_update_single<<<64, 64>>>(device_store);
  cavi_weighted_copy_back();
}
