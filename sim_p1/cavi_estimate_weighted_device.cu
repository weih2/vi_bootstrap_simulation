#include "hip/hip_runtime.h"
__device__
void cavi_implementation::device_generate_weights(){

}

__device__
void cavi_implementation::device_cavi_estimate_weighted(){

}

// for this method we only need vb posterior mean m_k
__global__
void cavi_implementation::device_cavi_bootstrap_update(){
  int tread_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(tread_id >= n_bootstrap) return;

  device_generate_weights(tread_id);

  double device_elbo;
  double device_old_elbo;
}
