#include "hip/hip_runtime.h"
void cavi_implementation::device_init_cavi_weighted(){
  // copy global setting to device
  hipMalloc((void**)&(device_g_vars.device_K), sizeof(int));
  hipMemcpy(device_g_vars.device_K, &data.g_vars.K, sizeof(int), hipMemcpyHostToDevice);
  hipMalloc((void**)&(device_g_vars.device_sigma_2), sizeof(double));
  hipMemcpy(device_g_vars.device_sigma_2, &data.g_vars.sigma_2, sizeof(double),
    hipMemcpyHostToDevice);
  hipMalloc((void**)&(device_g_vars.device_n_samples), sizeof(int));
  hipMemcpy(davice_g_vars.device_n_samples, &data.g_vars.n_samples, sizeof(int),
    hipMemcpyHostToDevice);

  hipMalloc((void**)&device_n_boostrap_samples, sizeof(int));
  hipMemcpy(device_n_boostrap_samples, &n_bootstrap_samples,
    sizeof(int), hipMemcpyHostToDevice);
  // allocate memory for weights using global memory temporarily
  hipMalloc((void**)&device_weights,
    n_bootstrap_samples * data.g_vars.n_samples * sizeof(double));
  // allocate memory for device data
  hipMalloc((void**)&device_x, data.g_vars.n_samples * sizeof(double));
  // allocate memory for device latent variables
  hipMalloc((void**)&(device_est.device_m),
    n_bootstrap_samples * data.g_vars.K * sizeof(double));
  hipMalloc((void**)&(device_est.device_s2),
    n_bootstrap_samples * data.g_vars.K * sizeof(double));
  hipMalloc((void**)&(device_est.device_phi),
    n_bootstrap_samples * data.g_vars.n_samples * data.g_vars.K * sizeof(double));
  // allocate another block of memory for rearranged latent variables
  hipMalloc((void**)&(device_est.device_m_transpose),
    n_bootstrap_samples * data.g_vars.K * sizeof(double));
  // allocate for device elbo
  hipMalloc((void**)(&device_elbo), n_bootstrap_samples * sizeof(double));

  // cavi settings
  hipMalloc((void**)(&device_epsilon), sizeof(double));
  hipMemcpy(device_epsilon, &epsilon, sizeof(dobule), hipMemcpyHostToDevice);
  hipMalloc((void**)(&device_max_n_iter), sizeof(int));
  hipMemcpy(device_max_n_iter, &max_n_iter, sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)(&device_exp_id), sizeof(int));
}

/*
The storage is in the spirit of
[n_bootstrap][n_sample][n_variable]
for all variables
except for device_est.device_m_transpose
*/

void cavi_implementation::cavi_weighted_copy_back(){
  // copy the latent variables back to ram
  hipMemcpy(host_m_transpose, device_est.device_m_transpose,
     n_bootstrap_samples * data.g_vars.K * sizeof(double),
     hipMemcpyDeviceToHost);
}

__device__
void cavi_implementation::device_generate_weights(int exp_id, int thread_id){
  // device_weights
  hiprandState state;
  hiprand_init(exp_id, thread_id, 0, &state);

  for(int n_sample = 0; n_sample < *device_n_boostrap_samples; n_sample++){
    device_weights[(*device_n_bootstrap_samples) * tread_id + n_sample]
      = hiprand_uniform(&state);
  }
}

__device__  // update estimate per thread per loop
void cavi_implementation::device_cavi_estimate_weighted(int thread_id){
  device_elbo[tread_id] = 0;
  double sum_phi;
  int phi_index;
  int par_index;
  int weight_index;

  for(int i = 0; i < *davice_g_vars.device_n_samples; i++){
    sum_phi = 0;
    weight_index = (*device_n_bootstrap_samples) * tread_id + i;

    for(int k = 0; k < *device_g_vars.device_K; k++){
      phi_index = tread_id
        * (*davice_g_vars.device_n_samples) * (*device_g_vars.device_K)
        + i
        * (*device_g_vars.device_K)
        + k;
      par_index = tread_id * (*device_g_vars.device_K) + k;

      sum_phi += (
        device_est.device_phi[par_index] =
          exp(device_x[i] * device_est.device_m[par_index]
            - (device_est.device_s2[par_index]
            + device_est.device_m[par_index]*device_est.device_m[par_index])/2.)
      );
    }
    for(int k = 0; k < *device_g_vars.device_K; k++){
      phi_index = tread_id
        * (*davice_g_vars.device_n_samples) * (*device_g_vars.device_K)
        + i
        * (*device_g_vars.device_K)
        + k;
      par_index = tread_id * (*device_g_vars.device_K) + k;

      device_est.device_phi[phi_index] /= sum_phi;
      device_elbo[tread_id] -=
        device_est.device_phi[phi_index]
        * device_weights[weight_index]
        * log(device_est.device_phi[phi_index]);
    }
  }

  double product_x_phi;
  for(int k = 0; k < *device_g_vars.device_K; k++){
    sum_phi = 0;
    product_x_phi = 0;

    par_index = tread_id * (*device_g_vars.device_K) + k;

    for(int i = 0; i < *davice_g_vars.device_n_samples; i++){
      phi_index = tread_id
        * (*davice_g_vars.device_n_samples) * (*device_g_vars.device_K)
        + i
        * (*device_g_vars.device_K)
        + k;
      weight_index = (*device_n_bootstrap_samples) * tread_id + i;
      sum_phi += device_est.device_phi[phi_index] * device_weights[weight_index];
      product_x_phi += device_x[i] * device_est.device_phi[phi_index]
        * device_weights[weight_index];
    }
    device_est.device_s2[par_index] = 1 / (1/(*device_g_vars.device_sigma_2) + sum_phi);
    device_est.device_m[par_index] = product_x_phi * device_est.device_s2[par_index];

    device_elbo[tread_id] +=
      - (product_x_phi * product_x_phi + 1) * device_est.device_s2[par_index]
        /(2*(*device_g_vars.device_sigma_2))
      + log(device_est.device_s2[par_index])/2.;
    for(int i = 0; i < data.g_vars.n_samples; i++){
      phi_index = tread_id
        * (*davice_g_vars.device_n_samples) * (*device_g_vars.device_K)
        + i
        * (*device_g_vars.device_K)
        + k;
      weight_index = (*device_n_bootstrap_samples) * tread_id + i;

      device_elbo[tread_id] += device_est.device_phi[phi_index] * device_weights[weight_index]
        *(device_x[i] * (-device_x[i]/2. + device_est.device_m[par_index])
        - (device_est.device_s2[par_index]
          + device_est.device_m[par_index]*device_est.device_m[par_index])/2.);
    }
  }
}

// for this method we only need vb posterior mean m_k
__global__
void cavi_implementation::device_cavi_bootstrap_update_single(){
  int tread_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(tread_id >= *device_n_boostrap_samples) return;

  const int par_index_start = tread_id * (*device_g_vars.device_K);
  const int par_index_end = (tread_id + 1) * (*device_g_vars.device_K);

  device_generate_weights((*device_exp_id), tread_id);
  double device_old_elbo;

  for(int n_iter = 0; n_iter < *device_max_n_iter; n_iter++){
    device_old_elbo = device_elbo[thread_id];
    device_cavi_estimate_weighted(tread_id);
    if(device_elbo[thread_id] - device_old_elbo < *device_epsilon) break;
  }

  thrust::sort_by_key(device_est.device_m + par_index_start,
    device_est.device_m + par_index_end,
    device_est.device_s2 + par_index_start);

  // rearrange
  for(int par_index = 0; par_index < (*device_g_vars.device_K); par_index++){
    device_est.device_m_transpose[+ tread_id]
      = device_est.device_m[par_index_start + par_index * (*device_n_boostrap_samples)];
  }
}

void cavi_implementation::device_cavi_bootstrap_update(){
  // yeah only one line
  hipMemcpy(device_exp_id, &n_experiments, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_x, data.x, data.g_vars.n_samples * sizeof(int) , hipMemcpyHostToDevice);
  device_cavi_bootstrap_update_single<<<64, 64>>>();
}
