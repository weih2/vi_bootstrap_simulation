#include "hip/hip_runtime.h"
void cavi_implementation::device_init_cavi_weighted(){
  // copy global setting to device
  hipMalloc((void**)&(device_g_vars.device_K), sizeof(int));
  hipMemcpy(device_g_vars.device_K, &data.g_vars.K, sizeof(int), hipMemcpyHostToDevice);
  hipMalloc((void**)&(device_g_vars.device_sigma_2), sizeof(double));
  hipMemcpy(device_g_vars.device_sigma_2, &data.g_vars.sigma_2, sizeof(double),
    hipMemcpyHostToDevice);
  hipMalloc((void**)&(device_g_vars.device_n_samples), sizeof(int));
  hipMemcpy(davice_g_vars.device_n_samples, &data.g_vars.n_samples, sizeof(int),
    hipMemcpyHostToDevice);

  hipMalloc((void**)&device_n_boostrap_samples, sizeof(int));
  hipMemcpy(device_n_boostrap_samples, &n_bootstrap_samples,
    sizeof(int), hipMemcpyHostToDevice);
  // allocate memory for weights using global memory temporarily
  hipMalloc((void**)&device_weights,
    n_bootstrap_samples * data.g_vars.n_samples * sizeof(double));
  // allocate memory for device data
  hipMalloc((void**)&device_x, data.g_vars.n_samples * sizeof(double));
  // allocate memory for device latent variables
  hipMalloc((void**)&(device_est.device_m),
    n_bootstrap_samples * data.g_vars.K * sizeof(double));
  hipMalloc((void**)&(device_est.device_s2),
    n_bootstrap_samples * data.g_vars.K * sizeof(double));
  hipMalloc((void**)&(device_est.device_phi),
    n_bootstrap_samples * data.g_vars.n_samples * data.g_vars.K * sizeof(double));
  // allocate another block of memory for rearranged latent variables
  hipMalloc((void**)&(device_est.device_m_transpose),
    n_bootstrap_samples * data.g_vars.K * sizeof(double));
}

/*
The storage is in the spirit of
[n_bootstrap][n_sample][n_variable]
for all variables
except for device_est.device_m_transpose
*/

void cavi_implementation::cavi_weighted_copy_back(){
  // copy the latent variables back to ram
}

__device__
void cavi_implementation::device_generate_weights(int exp_id, int thread_id){
  // device_weights
  hiprandState state;
  hiprand_init(exp_id, thread_id, 0, &state);

  for(int n_sample = 0; n_sample < *device_n_boostrap_samples; n_sample++){
    device_weights[(*device_n_bootstrap_samples) * tread_id + ] 
      = hiprand_uniform(&state);
  }
}

__device__
void cavi_implementation::device_cavi_estimate_weighted(){

}

// for this method we only need vb posterior mean m_k
__global__
void cavi_implementation::device_cavi_bootstrap_update_single(){
  int tread_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(tread_id >= n_bootstrap_samples) return;

  device_generate_weights(tread_id);

  double device_elbo;
  double device_old_elbo;

  for(){

  }
  thrust::sort_by_key();
  // rearrange
  for(){

  }
}

void cavi_implementation::device_cavi_bootstrap_update(){
  // yeah only one line
  device_cavi_bootstrap_update_single<<<64, 64>>>();
}
