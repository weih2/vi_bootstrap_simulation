#include "hip/hip_runtime.h"
// #define SEED_RUNIFORM 0

__device__ double device_random_uniform(int thread_id){
  hiprandState state;
  // seed = 12345
  hiprand_init(clock64(), thread_id, 0, &state);
  return hiprand_uniform(&state);
}
