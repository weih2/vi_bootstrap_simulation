// #define SEED_RUNIFORM 0

__device__
inline double device_random_uniform(int thread_id, int exp_id){
  hiprandState state;
  // seed = 12345
  // i hope this will work for each thread
  // that is, each thread has a private static inited indicator
  static int inited = 0;
  if(!inited){
    hiprand_init(exp_id, thread_id, 0, &state);
    inited = 1;
  }
  return hiprand_uniform(&state);
}
