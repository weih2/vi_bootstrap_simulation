#include "hip/hip_runtime.h"
__device__
void device_generate_weights(int exp_id, int thread_id, device_storage device_store){
  // device_store.device_weights
  hiprandState state;
  hiprand_init(exp_id, thread_id, 0, &state);

  for(int n_sample = 0; n_sample < *device_store.device_n_bootstrap_samples; n_sample++){
    device_store.device_weights[(*device_store.device_n_bootstrap_samples) * thread_id + n_sample]
      // = hiprand_uniform_double(&state);
      = 1;
  }
}

__device__  // update estimate per thread per loop
void device_cavi_estimate_weighted(int thread_id, device_storage device_store){
  device_store.device_elbo[thread_id] = 0;
  double sum_phi;
  int phi_index;
  int par_index;
  int weight_index;

  for(int i = 0; i < *device_store.device_g_vars.device_n_samples; i++){
    sum_phi = 0;
    weight_index = (*device_store.device_n_bootstrap_samples) * thread_id + i;

    for(int k = 0; k < *device_store.device_g_vars.device_K; k++){
      phi_index = thread_id
        * (*device_store.device_g_vars.device_n_samples) * (*device_store.device_g_vars.device_K)
        + i
        * (*device_store.device_g_vars.device_K)
        + k;
      par_index = thread_id * (*device_store.device_g_vars.device_K) + k;

      sum_phi += (
        (device_store.device_est.device_phi)[phi_index] =
          exp((device_store.device_x)[i] * (device_store.device_est.device_m)[par_index]
            - ((device_store.device_est.device_s2)[par_index]
            + (device_store.device_est.device_m)[par_index]*(device_store.device_est.device_m)[par_index])/2.)
      );
    }
    for(int k = 0; k < *device_store.device_g_vars.device_K; k++){
      phi_index = thread_id
        * (*device_store.device_g_vars.device_n_samples) * (*device_store.device_g_vars.device_K)
        + i
        * (*device_store.device_g_vars.device_K)
        + k;
      par_index = thread_id * (*device_store.device_g_vars.device_K) + k;

      device_store.device_est.device_phi[phi_index] /= sum_phi;
      device_store.device_elbo[thread_id] -=
        device_store.device_est.device_phi[phi_index]
        * device_store.device_weights[weight_index]
        * log(device_store.device_est.device_phi[phi_index]);
    }
  }

  double product_x_phi;
  for(int k = 0; k < *device_store.device_g_vars.device_K; k++){
    sum_phi = 0;
    product_x_phi = 0;

    par_index = thread_id * (*device_store.device_g_vars.device_K) + k;

    for(int i = 0; i < *device_store.device_g_vars.device_n_samples; i++){
      phi_index = thread_id
        * (*device_store.device_g_vars.device_n_samples) * (*device_store.device_g_vars.device_K)
        + i
        * (*device_store.device_g_vars.device_K)
        + k;
      weight_index = (*device_store.device_n_bootstrap_samples) * thread_id + i;
      sum_phi += device_store.device_est.device_phi[phi_index] * device_store.device_weights[weight_index];
      product_x_phi += device_store.device_x[i] * device_store.device_est.device_phi[phi_index]
        * device_store.device_weights[weight_index];
    }
    
    device_store.device_est.device_s2[par_index] = 1 / (1/(*device_store.device_g_vars.device_sigma_2) + sum_phi);
    device_store.device_est.device_m[par_index] = product_x_phi * device_store.device_est.device_s2[par_index];

    device_store.device_elbo[thread_id] +=
      - (product_x_phi * product_x_phi + 1) * device_store.device_est.device_s2[par_index]
        /(2*(*device_store.device_g_vars.device_sigma_2))
      + log(device_store.device_est.device_s2[par_index])/2.;
    for(int i = 0; i < *device_store.device_g_vars.device_n_samples; i++){
      phi_index = thread_id
        * (*device_store.device_g_vars.device_n_samples) * (*device_store.device_g_vars.device_K)
        + i
        * (*device_store.device_g_vars.device_K)
        + k;
      weight_index = (*device_store.device_n_bootstrap_samples) * thread_id + i;

      device_store.device_elbo[thread_id] += device_store.device_est.device_phi[phi_index] * device_store.device_weights[weight_index]
        *(device_store.device_x[i] * (-device_store.device_x[i]/2. + device_store.device_est.device_m[par_index])
        - (device_store.device_est.device_s2[par_index]
          + device_store.device_est.device_m[par_index]*device_store.device_est.device_m[par_index])/2.);

    }
  }
}

__global__
void device_cavi_bootstrap_update_single(device_storage device_store){
  int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(thread_id >= *device_store.device_n_bootstrap_samples) return;

  const int par_index_start = thread_id * (*device_store.device_g_vars.device_K);
  const int par_index_end = (thread_id + 1) * (*device_store.device_g_vars.device_K);

  device_generate_weights((*device_store.device_exp_id), thread_id, device_store);
  double device_old_elbo;

  for(int n_iter = 0; n_iter < *device_store.device_max_n_iter; n_iter++){
    device_old_elbo = device_store.device_elbo[thread_id];
    device_cavi_estimate_weighted(thread_id, device_store);
    if(device_store.device_elbo[thread_id] - device_old_elbo < *device_store.device_epsilon) break;
  }

  thrust::sort_by_key(thrust::device, device_store.device_est.device_m + par_index_start,
    device_store.device_est.device_m + par_index_end,
    device_store.device_est.device_s2 + par_index_start);

  // rearrange
  for(int par_index = 0; par_index < (*device_store.device_g_vars.device_K); par_index++){
    device_store.device_est.device_m_transpose[par_index * (*device_store.device_n_bootstrap_samples) + thread_id]
      = device_store.device_est.device_m[par_index_start + par_index];
  }
}
