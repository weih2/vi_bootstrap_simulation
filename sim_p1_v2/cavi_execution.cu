#include "hip/hip_runtime.h"
__global__ void cavi_execute(bridge bg){
  int t_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(t_id >= n_experiments) return;
  device_cavi_implementation thread_implementation(bg.device_dev_settings, t_id);
}
