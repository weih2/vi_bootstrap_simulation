#include "hip/hip_runtime.h"
__global__ void cavi_execute(bridge* bg){
  int t_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(t_id >= n_experiments) return;
  printf("my id is %d", t_id);
  device_cavi_implementation thread_implementation(bg->device_dev_settings, t_id);

  // obtain point estimates
  thread_implementation.device_cavi_point_estimate();
  for(int k = 0; k < K; k++){
    (bg -> device_empirical_mu)[t_id + k * n_experiments] = thread_implementation.m[k];
  }
}
