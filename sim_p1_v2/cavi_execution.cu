#include "hip/hip_runtime.h"
__global__ void cavi_execute(bridge bg, int bootstrap_execution){
  int t_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(t_id >= n_experiments) return;
  device_cavi_implementation thread_implementation(bg.device_dev_settings, t_id);
  // obtain point estimates
  thread_implementation.device_cavi_point_estimate();

  if(bootstrap_execution != 0){
    thread_implementation.device_weighted_cavi_point_estimate();
    thread_implementation.device_vwlb_cs_construct();
    // thread_implementation.device_vwlb_cs2_construct();
    thread_implementation.device_vp_cs_construct();

    /*
    bg.dev_vp_first_centers[t_id] = cal_mean(thread_implementation.vp_cs[0], 2);
    bg.dev_vwlb_first_centers[t_id] = cal_mean(thread_implementation.vwlb_cs[0], 2);
    bg.dev_vwlb2_first_centers[t_id] = cal_mean(thread_implementation.vwlb_cs2[0], 2);
    */

    for(int k = 0; k < N_CLUSTERS; k++){
      if((thread_implementation.vp_cs[k][0] < thread_implementation.mu[k])
        &&(thread_implementation.vp_cs[k][1] > thread_implementation.mu[k]))
        bg.device_vp_cs_covered[k * n_experiments + t_id] = 1;
        else bg.device_vp_cs_covered[k * n_experiments + t_id] = 0;
      if((thread_implementation.vwlb_cs[k][0] < thread_implementation.mu[k])
        &&(thread_implementation.vwlb_cs[k][1] > thread_implementation.mu[k]))
        bg.device_vwlb_cs_covered[k * n_experiments + t_id] = 1;
        else bg.device_vwlb_cs_covered[k * n_experiments + t_id] = 0;
      if((thread_implementation.vwlb_cs2[k][0] < thread_implementation.mu[k])
        &&(thread_implementation.vwlb_cs2[k][1] > thread_implementation.mu[k]))
        bg.device_vwlb_cs2_covered[k * n_experiments + t_id] = 1;
        else bg.device_vwlb_cs2_covered[k * n_experiments + t_id] = 0;
    }
  }

  // bg.device_is_outlier[t_id] = thread_implementation.is_outlier;

  if(bootstrap_execution == 2){  // consider length
    for(int k = 0; k < N_CLUSTERS; k++){
      bg.device_vwlb_cs_lengths[k * n_experiments + t_id] =
        thread_implementation.vwlb_cs[k][1] - thread_implementation.vwlb_cs[k][0];
      bg.device_vwlb_cs2_lengths[k * n_experiments + t_id] =
        thread_implementation.vwlb_cs2[k][1] - thread_implementation.vwlb_cs2[k][0];
      bg.device_vp_cs_lengths[k * n_experiments + t_id] =
        thread_implementation.vp_cs[k][1] - thread_implementation.vp_cs[k][0];
    }
  }

  // in a non-bootstrap implementation this is all we need
  for(int k = 0; k < N_CLUSTERS; k++){
    (bg.device_empirical_mu)[t_id + k * n_experiments] = thread_implementation.m[k];
  }
}
