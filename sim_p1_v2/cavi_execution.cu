#include "hip/hip_runtime.h"
__global__ void cavi_execute(bridge bg){
  int t_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(t_id >= n_experiments) return;
  device_cavi_implementation thread_implementation(bg.device_dev_settings, t_id);
  // obtain point estimates
  thread_implementation.device_cavi_point_estimate();

  thread_implementation.device_weighted_cavi_point_estimate();
  thread_implementation.device_vwlb_cs_construct();
  thread_implementation.device_vp_cs_construct();

  for(int k = 0; k < K; k++){
    (bg.device_empirical_mu)[t_id + k * n_experiments] = thread_implementation.m[k];
    if(t_id == 0){
      printf("ci is [%f,%f]\n", thread_implementation.vwlb_cs[k][0],
       thread_implementation.vwlb_cs[k][1]);
    }
  }
}
