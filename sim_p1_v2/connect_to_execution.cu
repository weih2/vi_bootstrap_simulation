void bridge::connect_to_execution(){
  cavi_execute<<<64,64>>>(*this, 1);
  hipDeviceSynchronize();
  for(int n = 0; n < n_experiments; n++){
    cavi_execute<<<64,64>>>(*this, 0);
    hipDeviceSynchronize();
    construct_empirical_ci();

    for(int k = 0; k < K; k++){
      if((empirical_ci[k][0] < host_dev_settings.l_vars.mu[k])
        &&(empirical_ci[k][1] > host_dev_settings.l_vars.mu[k]))
        empirical_ci_covered_counts[k]++;
    }
  }
}
