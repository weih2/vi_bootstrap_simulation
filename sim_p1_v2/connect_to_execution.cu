void bridge::connect_to_execution(){
  cavi_execute<<<64,64>>>(*this, 1);
  hipDeviceSynchronize();
  count_coverage();
  hipDeviceSynchronize();
}

void bridge::connect_to_analysis(){
  cavi_execute<<<64,64>>>(*this, 2);
  hipDeviceSynchronize();
  count_coverage();
  lengths_copy_back();
  hipDeviceSynchronize();
}
