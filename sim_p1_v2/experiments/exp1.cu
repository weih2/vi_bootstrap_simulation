#include "hip/hip_runtime.h"
#include "../include.h"
#include "fixed_latent_vars_generation.cpp"
#define DELTA_COUNT 51

int main(){
    latent_vars true_vars;
    generate_latent_pars(true_vars);

    device_settings dev_settings;
    dev_settings.l_vars = true_vars;

    dev_settings.epsilon = new double();
    *dev_settings.epsilon = 0.001;

    dev_settings.max_n_iter = new int();
    *dev_settings.max_n_iter = 100;

    dev_settings.bootstrap_confidence = new double();
    *dev_settings.bootstrap_confidence = 0.95;

    dev_settings.ci_quantile = new double();
    *dev_settings.ci_quantile = cdf_ugaussian_Pinv(0.975);

    double vwlb_cs_covered_counts_total[DELTA_COUNT];
    double vp_cs_covered_counts_total[DELTA_COUNT];
    double empirical_ci_covered_counts_total[DELTA_COUNT];
    double vwlb_cs2_covered_counts_total[DELTA_COUNT];

    int n_outliers[DELTA_COUNT];

    bridge bridge_settings(dev_settings);
    bridge_settings.save_settings(std::cout);
    bridge_settings.clean_device();

    for(int delta_count = 0; delta_count < DELTA_COUNT; delta_count++){
      fixed_latent_vars_generation(dev_settings.l_vars,
        exp(-3 + delta_count * 0.1));

      bridge bridge_0(dev_settings);

      bridge_0.connect_to_execution();

      for(int k = 0; k < K; k++){
        vwlb_cs_covered_counts_total[delta_count] = 0;
        vp_cs_covered_counts_total[delta_count] = 0;
        empirical_ci_covered_counts_total[delta_count] = 0;
        vwlb_cs2_covered_counts_total[delta_count] = 0;
      }

      n_outliers[delta_count] = bridge_0.n_outliers;

      for(int k = 0; k < K; k++){
        vwlb_cs_covered_counts_total[delta_count]
         += bridge_0.vwlb_cs_covered_counts[k];
        vp_cs_covered_counts_total[delta_count]
         += bridge_0.vp_cs_covered_counts[k];
       vwlb_cs2_covered_counts_total[delta_count]
         += bridge_0.vwlb_cs2_covered_counts[k];
        empirical_ci_covered_counts_total[delta_count]
         += bridge_0.empirical_ci_covered_counts[k];
      }

      bridge_0.clean_device();
      printf("experiments #%d is done\n", delta_count);
    }

    printf("number of outliers in each experiment: \n");

    for(int delta_count = 0; delta_count < DELTA_COUNT; delta_count++)
      printf("%d ", n_outliers[delta_count]);
    printf("\n");

    int n_effective;
    for(int delta_count = 0; delta_count < DELTA_COUNT; delta_count++){
      n_effective = n_experiments - n_outliers[delta_count];
      vwlb_cs_covered_counts_total[delta_count] /= double(K * n_effective);
      vp_cs_covered_counts_total[delta_count] /= double(K * n_effective);
      empirical_ci_covered_counts_total[delta_count] /= double(K * n_effective);
      vwlb_cs2_covered_counts_total[delta_count] /= double(K * n_effective);
      printf("%f ", vwlb_cs_covered_counts_total[delta_count]);
    }
    printf("\n");

    for(int delta_count = 0; delta_count < DELTA_COUNT; delta_count++){
      printf("%f ", vwlb_cs2_covered_counts_total[delta_count]);
    }

    printf("\n");

    for(int delta_count = 0; delta_count < DELTA_COUNT; delta_count++){
      printf("%f ", vp_cs_covered_counts_total[delta_count]);
    }

    printf("\n");

    for(int delta_count = 0; delta_count < DELTA_COUNT; delta_count++){
      printf("%f ", empirical_ci_covered_counts_total[delta_count]);
    }

  return 0;
}
