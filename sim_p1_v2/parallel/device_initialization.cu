#include "hip/hip_runtime.h"
__device__ device_cavi_implementation::device_cavi_implementation(device_settings dev_settings, int t_id){
  // copy local variables
  for(int k = 0; k < K; k++){
    mu[k] = dev_settings.l_vars.mu[k];
  }

  for(int i = 0; i < n_samples; i++){
    c[i] = dev_settings.l_vars.c[i];
  }

  // copy settings
  device_max_n_iter = *dev_settings.max_n_iter;
  device_epsilon = *dev_settings.epsilon;
  device_bootstrap_confidence = *dev_settings.bootstrap_confidence;
  device_ci_quantile = *dev_settings.ci_quantile;

  // generate data
  thread_id = t_id;
  b_count = 0;

  // set random state
  hiprandState state;
  hiprand_init(thread_id + n_experiments * (*dev_settings.data_count), 0, 0, &state);
  if(t_id == 0) (*dev_settings.data_count)++;

  // initialize estimates
  for(int k = 0; k < K; k++){
    m[k] = hiprand_normal_double(&state) * sqrt(double(sigma_2));
    s2[k] = sigma_2;
    for(int i = 0; i < n_samples; i++){
      phi[i][k] = 1/double(K);
    }
  }

  for(int i = 0; i < n_samples; i++){
    // let variance be 1
    x[i] = mu[c[i]] + hiprand_normal_double(&state);
  }
}
