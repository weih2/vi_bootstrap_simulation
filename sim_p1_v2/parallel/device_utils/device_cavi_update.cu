#include "hip/hip_runtime.h"
__device__ void device_cavi_implementation::device_cavi_point_estimate_update(){
  // setting constant
  elbo = 0;
  // update phi
  double sum_phi;
  for(int i = 0; i < n_samples; i++){
    sum_phi = 0;
    for(int k = 0; k < N_CLUSTERS; k++){
      phi[i][k] = exp(x[i] * m[k] - (s2[k] + m[k]*m[k])/2.);
      sum_phi += phi[i][k];
    }
    for(int k = 0; k < N_CLUSTERS; k++){
      phi[i][k] /= sum_phi;
      elbo -= phi[i][k] * log(phi[i][k]);
    }
  }

  // update posterior of mu
  double product_x_phi;
  for(int k = 0; k < N_CLUSTERS; k++){
    sum_phi = 0;
    product_x_phi = 0;

    for(int i = 0; i < n_samples; i++){
      sum_phi += phi[i][k];
      product_x_phi += x[i] * phi[i][k];
    }
    s2[k] = 1 / (1/double(sigma_2) + sum_phi);
    m[k] = product_x_phi * s2[k];

    elbo += - (product_x_phi * product_x_phi + 1) * s2[k]/(2.*double(sigma_2)) + log(s2[k])/2.;
    for(int i = 0; i < n_samples; i++){
      elbo += phi[i][k] *
        (x[i] * (- x[i]/2. + m[k]) - (s2[k] + m[k]*m[k])/2.);
    }
  }
}
