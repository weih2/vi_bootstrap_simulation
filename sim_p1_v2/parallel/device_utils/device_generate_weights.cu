#include "hip/hip_runtime.h"
__device__ void device_cavi_implementation::device_generate_weights(){
  hiprandState state;
  hiprand_init(thread_id, b_count + 1, 0, &state);

  double sum_weights = 0;
  for(int i = 0; i < n_samples; i++){
    sum_weights += (
      weights[i] = -log(hiprand_uniform_double(&state))
    );
  }

  for(int i = 0; i < n_samples; i++){
    weights[i] /= sum_weights;
  }

  // by the way we can initialize estimates
  for(int k = 0; k < K; k++){
    m_b[k] = hiprand_normal_double(&state) * sqrt(double(sigma_2));
    s2_b[k] = sigma_2;
    for(int i = 0; i < n_samples; i++){
      phi_b[i][k] = 1/double(K);
    }
  }
}
