#include "hip/hip_runtime.h"
__device__ void device_cavi_implementation::device_generate_weights(){
  hiprandState state;
  hiprand_init(thread_id, b_count, 0, &state);

  double sum_weights = 0;
  for(int i = 0; i < n_samples; i++){
    sum_weights += (
      weights[i] = -log(hiprand_uniform_double(&state))
    );
  }

  for(int i = 0; i < n_samples; i++){
    weights[i] /= sum_weights;
  }

  // by the way we can initialize estimates
  for(int k = 0; k < g_vars.K; k++){
    m_b[k] = hiprand_normal_double(&state) * sqrt(g_vars.sigma_2);
    s2_b[k] = sigma_2;
    for(int i = 0; i < n_samples; i++){
      phi_b[k] = 1/double(K);
    }
  }
}
