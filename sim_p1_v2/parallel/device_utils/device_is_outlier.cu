#include "hip/hip_runtime.h"
__device__ void device_cavi_implementation::device_is_outlier(){
  double b_sample_sds[N_CLUSTERS];
  n_outliers = 0;

  for(int k = 0; k < N_CLUSTERS; k++){
    b_sample_sds[k] = sqrt(cal_variance(map_mu[k], n_bootstrap_samples));
  }

  // is_outlier = 1;
  for(int b = 0; b < n_bootstrap_samples; b++){
    is_outlier[b] = 0;
    for(int k = 0; k < N_CLUSTERS; k++){
      if(0){
        is_outlier[b] = 1;
        n_outliers ++;
        break;
      }
    }
  }

  int b_clean = 0;
  for(int b = 0; b < n_bootstrap_samples; b++){
    if(!is_outlier[b]){
      for(int k = 0; k < N_CLUSTERS; k++){
        map_mu_clean[k][b_clean] = map_mu[k][b];
      }
      b_clean ++;
    }
  }

  return;
}
