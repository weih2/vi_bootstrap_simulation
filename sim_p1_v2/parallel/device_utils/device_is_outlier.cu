#include "hip/hip_runtime.h"
__device__ void device_cavi_implementation::device_is_outlier(){
  double b_sample_sds[K];
  n_outliers = 0;

  for(int k = 0; k < K; k++){
    b_sample_sds[k] = sqrt(cal_variance(map_mu[k], n_bootstrap_samples));
  }

  // is_outlier = 1;
  for(int b = 0; b < n_bootstrap_samples; b++){
    is_outlier[b] = 0;
    for(int k = 0; k < K; k++){
      if(fabs(map_mu[k][b] - m[k]) > 5 * b_sample_sds[k]){
        is_outlier[b] = 1;
        n_outliers ++;
        break;
      }
    }
  }

  int b_clean = 0;
  for(int b = 0; b < n_bootstrap_samples; b++){
    if(!is_outlier[b]){
      for(int k = 0; k < K; k++){
        map_mu_clean[k][b_clean] = map_mu[k][b];
      }
      b_clean ++;
    }
  }

  return;
}
