#include "hip/hip_runtime.h"
__device__ void device_cavi_implementation::device_is_outlier(){
  // detect inaccurate point estimate
  double absolute_deviance = 0;
  for(int k = 0; k < K; k++){
    absolute_deviance += fabs(m[k] - mu[k]);
  }

  if(absolute_deviance > 1){
    is_outlier = 1;
  }else is_outlier = 0;

  // is_outlier = 1;

  return;
}
