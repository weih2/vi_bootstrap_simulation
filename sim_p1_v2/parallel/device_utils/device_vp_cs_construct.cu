#include "hip/hip_runtime.h"
void device_cavi_implementation::device_vp_cs_construct(){
  for(int k = 0; k < K; k++){
    vp_cs[k][0] = -z_quantile * sqrt(s2[k]) + m[k];
    vp_cs[k][1] = z_quantile * sqrt(s2[k]) + m[k];
  }
}
