#include "hip/hip_runtime.h"
__device__ void device_cavi_implementation::device_vp_cs_construct(){
  for(int k = 0; k < K; k++){
    vp_cs[k][0] = -device_ci_quantile * sqrt(s2[k]) + m[k];
    vp_cs[k][1] = device_ci_quantile * sqrt(s2[k]) + m[k];
  }
}
