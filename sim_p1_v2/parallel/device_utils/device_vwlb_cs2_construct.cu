#include "hip/hip_runtime.h"
__device__ void device_cavi_implementation::device_vwlb_cs2_construct(){
  double b_sample_variance;

  for(int k = 0; k < K; k++){
    b_sample_variance = cal_variance(map_mu[k], n_bootstrap_samples);
    vwlb_cs2[k][0] = m[k] - device_ci_quantile * sqrt(b_sample_variance);
    vwlb_cs2[k][1] = m[k] + device_ci_quantile * sqrt(b_sample_variance);
  }
  
  if(thread_id == 0){
    printf("the bootstrap sample for first map is\n", );
    for(int n_b = 0; n_b < n_bootstrap_samples; n_b++){
      printf("%f ", map_mu[0][n_b]);
    }
    printf("\n");
  }
}
