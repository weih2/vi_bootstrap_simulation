#include "hip/hip_runtime.h"
__device__ void
device_cavi_implementation::device_weighted_cavi_point_estimate_update(){
  // setting constant
  elbo = 0;
  // update phi
  double sum_phi;
  for(int i = 0; i < n_samples; i++){
    sum_phi = 0;
    for(int k = 0; k < K; k++){
      phi_b[i][k] = exp(x[i] * m_b[k] - (s2_b[k] + m_b[k]*m_b[k])/2.);
      sum_phi += phi_b[i][k];
    }
    for(int k = 0; k < K; k++){
      phi_b[i][k] /= sum_phi;
      elbo -= phi_b[i][k] * log(phi_b[i][k]) * weights[i];
    }
  }

  // update posterior of mu
  double product_x_phi;
  for(int k = 0; k < K; k++){
    sum_phi = 0;
    product_x_phi = 0;

    for(int i = 0; i < n_samples; i++){
      sum_phi += phi_b[i][k] * weights[i];
      product_x_phi += x[i] * phi_b[i][k]* weights[i];
    }
    s2_b[k] = 1 / (1/double(sigma_2) + sum_phi);
    m_b[k] = product_x_phi * s2_b[k];

    elbo += - (product_x_phi * product_x_phi + 1) * s2_b[k]/(2.*double(sigma_2)) + log(s2_b[k])/2.;
    for(int i = 0; i < n_samples; i++){
      elbo += phi_b[i][k] * weights[i] *
        (x[i] * (- x[i]/2. + m_b[k]) - (s2_b[k] + m_b[k]*m_b[k])/2.);
    }
  }
}
