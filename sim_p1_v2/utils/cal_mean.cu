
#include <hip/hip_runtime.h>
__device__ __host__ inline double cal_mean(const double *observations, int n_observations){
  double mean = 0;
  for(int o = 0; o < n_observations; o++){
    mean += observations[o];
  }
  mean /= double(n_observations);
  return mean;
}

__device__ __host__ inline double cal_variance(const double *observations, int n_observations){
  double mean = cal_mean(observations, n_observations);

  double variance = 0;
  for(int o = 0; o < n_observations; o++){
    variance += (observations[o] - mean) * (observations[o] - mean);
  }
  variance /= double(n_observations - 1);
  return variance;
}
