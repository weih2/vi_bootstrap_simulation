#include "hip/hip_runtime.h"
void bridge::construct_empirical_ci(){
  hipMemcpy(host_empirical_mu, device_empirical_mu,
    sizeof(double) * K * n_experiments, hipMemcpyDeviceToHost);

  // estimate standard deviation
  double sum_m[K];
  double variance_m_est[K];
  for(int k = 0; k < K; k++){
    sum_m[k] = 0;
    variance_m_est[k] = 0;
    for(int n = 0; n < n_experiments; n++)
      sum_m[k] += host_empirical_mu[k*n_experiments + n];
    for(int n = 0; n < n_experiments; n++)
      variance_m_est[k] +=
        (host_empirical_mu[k*n_experiments + n] - sum_m[k]/double(n_experiments))
        *
        (host_empirical_mu[k*n_experiments + n] - sum_m[k]/double(n_experiments));
    variance_m_est[k] /= double(n_experiments - 1);

    empirical_ci[k][0] = host_dev_settings.l_vars.mu[k] -
      (*host_dev_settings.ci_quantile) * sqrt(variance_m_est[k]);
    empirical_ci[k][1] = host_dev_settings.l_vars.mu[k] +
      (*host_dev_settings.ci_quantile) * sqrt(variance_m_est[k]);
  }
}
