#include "hip/hip_runtime.h"
void bridge::construct_empirical_ci(){
  hipMemcpy(host_empirical_mu, device_empirical_mu,
    sizeof(double) * K * n_experiments, hipMemcpyDeviceToHost);

  hipMemcpy(is_outlier, device_is_outlier,
    sizeof(int) * n_experiments, hipMemcpyDeviceToHost);

  for(int n = 0; n < n_experiments; n++) n_outlier += is_outlier[n];

  // estimate standard deviation
  double sum_m[K];
  double variance_m_est[K];
  for(int k = 0; k < K; k++){
    sum_m[k] = 0;
    variance_m_est[k] = 0;
    for(int n = 0; n < n_experiments; n++){
      if(is_outlier[n]) continue;
      sum_m[k] += host_empirical_mu[k*n_experiments + n];
    }

    for(int n = 0; n < n_experiments; n++){
      if(is_outlier[n]) continue;
      variance_m_est[k] +=
        (host_empirical_mu[k*n_experiments + n] - sum_m[k]/double(n_experiments))
        *
        (host_empirical_mu[k*n_experiments + n] - sum_m[k]/double(n_experiments));
    }

    variance_m_est[k] /= double(n_experiments - n_outlier - 1);

    empirical_ci[k][0] = host_dev_settings.l_vars.mu[k] -
      (*host_dev_settings.ci_quantile) * sqrt(variance_m_est[k]);
    empirical_ci[k][1] = host_dev_settings.l_vars.mu[k] +
      (*host_dev_settings.ci_quantile) * sqrt(variance_m_est[k]);
  }
}
