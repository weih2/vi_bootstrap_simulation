#include "hip/hip_runtime.h"
void bridge::construct_empirical_ci(){
  hipMemcpy(host_empirical_mu, device_empirical_mu,
    sizeof(double) * K * n_experiments, hipMemcpyDeviceToHost);

  // hipMemcpy(is_outlier, device_is_outlier,
  //  sizeof(int) * n_experiments, hipMemcpyDeviceToHost);

  // estimate standard deviation
  double avg_m[K];
  double sd_m_est[K];
  for(int k = 0; k < K; k++){
    avg_m[k] = 0;
    sd_m_est[k] = 0;
    for(int n = 0; n < n_experiments; n++){
      avg_m[k] += host_empirical_mu[k*n_experiments + n];
    }

    avg_m[k] /= double(n_experiments);

    for(int n = 0; n < n_experiments; n++){
      sd_m_est[k] +=
        (host_empirical_mu[k*n_experiments + n] - avg_m[k])
        *
        (host_empirical_mu[k*n_experiments + n] - avg_m[k]);
    }

    sd_m_est[k] /= double(n_experiments - 1);
    sd_m_est[k] = sqrt(sd_m_est[k]);

  }
    // detect outlier
  for(int n = 0; n < n_experiments; n++){
    is_outlier[n] = 0;
    for(int k = 0; k < K; k++){
      if(fabs(host_empirical_mu[k*n_experiments + n] - avg_m[k]) > sd_m_est[k] * 3){
        continue;
        is_outlier[n] = 1;
        n_outliers ++;
        continue;
      }
    }
  }

  for(int k = 0; k < K; k++){
    avg_m[k] = 0;
    sd_m_est[k] = 0;
    for(int n = 0; n < n_experiments; n++){
      if(is_outlier[n]) continue;
      avg_m[k] += host_empirical_mu[k*n_experiments + n];
    }

    avg_m[k] /= double(n_experiments);

    for(int n = 0; n < n_experiments; n++){
      if(is_outlier[n]) continue;
      sd_m_est[k] +=
        (host_empirical_mu[k*n_experiments + n] - avg_m[k])
        *
        (host_empirical_mu[k*n_experiments + n] - avg_m[k]);
    }

    sd_m_est[k] /= double(n_experiments - n_outliers - 1);
    sd_m_est[k] = sqrt(sd_m_est[k]);

      empirical_ci[k][0] = host_dev_settings.l_vars.mu[k] -
        (*host_dev_settings.ci_quantile) * sd_m_est[k];
      empirical_ci[k][1] = host_dev_settings.l_vars.mu[k] +
        (*host_dev_settings.ci_quantile) * sd_m_est[k];
  }
}
