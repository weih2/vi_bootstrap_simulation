void bridge::copy_back(){
  // waiting for execution of threads
  hipDeviceSynchronize();

  hipMemcpy(host_empirical_mu, device_empirical_mu,
    sizeof(double) * K * n_experiments, hipMemcpyDeviceToHost);

  hipMemcpy(vwlb_cs_covered, device_vwlb_cs_covered,
    sizeof(int) * K * n_experiments, hipMemcpyDeviceToHost);

  hipMemcpy(vp_cs_covered, device_vp_cs_covered,
    sizeof(int) * K * n_experiments, hipMemcpyDeviceToHost);
}
