void bridge::copy_back(){
  hipMemcpy(host_empirical_mu, device_empirical_mu,
    sizeof(double) * K * n_experiments, hipMemcpyDeviceToHost);

  hipMemcpy(vwlb_cs_covered, device_vwlb_cs_covered,
    sizeof(int) * K * n_experiments, hipMemcpyDeviceToHost);

  hipMemcpy(vp_cs_covered, device_vp_cs_covered,
    sizeof(int) * K * n_experiments, hipMemcpyDeviceToHost);
}
