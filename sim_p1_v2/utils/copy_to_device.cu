void bridge::init_device(){
  // allocate memory
  hipMalloc((void**)&device_dev_settings.l_vars.mu, g_vars.K * sizeof(double));
  hipMalloc((void**)&device_dev_settings.l_vars.c, g_vars.n_samples * sizeof(int));
  hipMalloc((void**)&device_dev_settings.epsilon, sizeof(double));
  hipMalloc((void**)&device_dev_settings.max_n_iter, sizeof(int));
  hipMalloc((void**)&device_dev_settings.bootstrap_confidence, sizeof(double));
  hipMalloc((void**)&device_dev_settings.ci_quantile, sizeof(double));

  hipMalloc((void**)&device_vwlb_cs_covered,
    sizeof(int) * g_vars.K * n_bootstrap_samples);
  hipMalloc((void**)&device_vp_cs_covered,
    sizeof(int) * g_vars.K * n_bootstrap_samples);
}

void bridge::copy_to_device(){
  hipMemcpy(device_dev_settings.l_vars.mu, host_dev_settings.l_vars.mu,
    g_vars.K * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.l_vars.c, host_dev_settings.l_vars.c,
    g_vars.n_samples * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.epsilon, host_dev_settings.epsilon,
    sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.max_n_iter, host_dev_settings.max_n_iter,
    sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.bootstrap_confidence, host_dev_settings.bootstrap_confidence,
    sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.ci_quantile, host_dev_settings.ci_quantile,
    sizeof(double), hipMemcpyHostToDevice);
}
