void bridge::init_device(){
  // allocate memory
  hipMalloc((void**)&device_dev_settings.l_vars.mu, N_CLUSTERS * sizeof(double));
  hipMalloc((void**)&device_dev_settings.l_vars.c, n_samples * sizeof(int));
  hipMalloc((void**)&device_dev_settings.epsilon, sizeof(double));
  hipMalloc((void**)&device_dev_settings.max_n_iter, sizeof(int));
  hipMalloc((void**)&device_dev_settings.bootstrap_confidence, sizeof(double));
  hipMalloc((void**)&device_dev_settings.ci_quantile, sizeof(double));

  hipMalloc((void**)&device_dev_settings.data_count, sizeof(int));

  hipMalloc((void**)&device_vwlb_cs_covered,
    sizeof(int) * N_CLUSTERS * n_experiments);
  hipMalloc((void**)&device_vwlb_cs2_covered,
    sizeof(int) * N_CLUSTERS * n_experiments);
  hipMalloc((void**)&device_vp_cs_covered,
    sizeof(int) * N_CLUSTERS * n_experiments);

  hipMalloc((void**)&device_vp_cs_lengths,
    sizeof(double) * n_experiments);
  hipMalloc((void**)&device_vwlb_cs_lengths,
    sizeof(double) * n_experiments);
  hipMalloc((void**)&device_vwlb_cs2_lengths,
    sizeof(double) * n_experiments);

  hipMalloc((void**)&device_empirical_mu,
    sizeof(double) * n_experiments * N_CLUSTERS);

  // hipMalloc((void**)&device_is_outlier,
  //  sizeof(int) * n_experiments);
  hipMalloc((void**)&dev_vp_first_centers, sizeof(double) * n_experiments * N_CLUSTERS);
  hipMalloc((void**)&dev_vwlb_first_centers, sizeof(double) * n_experiments * N_CLUSTERS);
  hipMalloc((void**)&dev_vwlb2_first_centers, sizeof(double) * n_experiments * N_CLUSTERS);
}

void bridge::clean_device(){
  hipFree(device_dev_settings.l_vars.mu);
  hipFree(device_dev_settings.l_vars.c);
  hipFree(device_dev_settings.epsilon);
  hipFree(device_dev_settings.max_n_iter);
  hipFree(device_dev_settings.bootstrap_confidence);
  hipFree(device_dev_settings.ci_quantile);
  hipFree(device_dev_settings.data_count);
  hipFree(device_vwlb_cs_covered);
  hipFree(device_vwlb_cs2_covered);
  hipFree(device_vp_cs_lengths);
  hipFree(device_vwlb_cs_lengths);
  hipFree(device_vwlb_cs2_lengths);
  hipFree(device_vp_cs_covered);
  hipFree(device_vp_cs_covered);
  hipFree(device_empirical_mu);
  // hipFree(device_is_outlier);
  hipFree(dev_vp_first_centers);
  hipFree(dev_vwlb_first_centers);
  hipFree(dev_vwlb2_first_centers);
}

void bridge::copy_to_device(){
  hipMemcpy(device_dev_settings.l_vars.mu, host_dev_settings.l_vars.mu,
    N_CLUSTERS * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.l_vars.c, host_dev_settings.l_vars.c,
    n_samples * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.epsilon, host_dev_settings.epsilon,
    sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.max_n_iter, host_dev_settings.max_n_iter,
    sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.bootstrap_confidence, host_dev_settings.bootstrap_confidence,
    sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.ci_quantile, host_dev_settings.ci_quantile,
    sizeof(double), hipMemcpyHostToDevice);
}
