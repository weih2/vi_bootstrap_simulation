void bridge::init_device(){
  // allocate memory
  hipMalloc((void**)&device_dev_settings.l_vars.mu, K * sizeof(double));
  hipMalloc((void**)&device_dev_settings.l_vars.c, n_samples * sizeof(int));
  hipMalloc((void**)&device_dev_settings.epsilon, sizeof(double));
  hipMalloc((void**)&device_dev_settings.max_n_iter, sizeof(int));
  hipMalloc((void**)&device_dev_settings.bootstrap_confidence, sizeof(double));
  hipMalloc((void**)&device_dev_settings.ci_quantile, sizeof(double));

  hipMalloc((void**)&device_vwlb_cs_covered,
    sizeof(int) * K * n_bootstrap_samples);
  hipMalloc((void**)&device_vp_cs_covered,
    sizeof(int) * K * n_bootstrap_samples);

  hipMalloc((void**)&device_empirical_mu,
    sizeof(double) * n_experiments * K);
}

void bridge::copy_to_device(){
  hipMemcpy(device_dev_settings.l_vars.mu, host_dev_settings.l_vars.mu,
    K * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.l_vars.c, host_dev_settings.l_vars.c,
    n_samples * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.epsilon, host_dev_settings.epsilon,
    sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.max_n_iter, host_dev_settings.max_n_iter,
    sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.bootstrap_confidence, host_dev_settings.bootstrap_confidence,
    sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.ci_quantile, host_dev_settings.ci_quantile,
    sizeof(double), hipMemcpyHostToDevice);
}
