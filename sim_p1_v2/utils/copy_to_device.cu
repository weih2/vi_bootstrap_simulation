void bridge::init_device(){
  // allocate memory
  hipMalloc((void**)&device_dev_settings.l_vars.mu, K * sizeof(double));
  hipMalloc((void**)&device_dev_settings.l_vars.c, n_samples * sizeof(int));
  hipMalloc((void**)&device_dev_settings.epsilon, sizeof(double));
  hipMalloc((void**)&device_dev_settings.max_n_iter, sizeof(int));
  hipMalloc((void**)&device_dev_settings.bootstrap_confidence, sizeof(double));
  hipMalloc((void**)&device_dev_settings.ci_quantile, sizeof(double));

  hipMalloc((void**)&device_dev_settings.data_count, sizeof(int));

  hipMalloc((void**)&device_vwlb_cs_covered,
    sizeof(int) * K * n_experiments);
  hipMalloc((void**)&device_vwlb_cs2_covered,
    sizeof(int) * K * n_experiments);
  hipMalloc((void**)&device_vp_cs_covered,
    sizeof(int) * K * n_experiments);

  hipMalloc((void**)&device_vp_cs_lengths,
    sizeof(double) * K * n_experiments);
  hipMalloc((void**)&device_vwlb_cs_lengths,
    sizeof(double) * K * n_experiments);
  hipMalloc((void**)&device_vwlb_cs2_lengths,
    sizeof(double) * K * n_experiments);

  hipMalloc((void**)&device_empirical_mu,
    sizeof(double) * n_experiments * K);
}

void bridge::clean_device(){
  hipFree(device_dev_settings.l_vars.mu);
  hipFree(device_dev_settings.l_vars.c);
  hipFree(device_dev_settings.epsilon);
  hipFree(device_dev_settings.max_n_iter);
  hipFree(device_dev_settings.bootstrap_confidence);
  hipFree(device_dev_settings.ci_quantile);
  hipFree(device_dev_settings.data_count);
  hipFree(device_vwlb_cs_covered);
  hipFree(device_vwlb_cs2_covered);
  hipFree(device_vp_cs_lengths);
  hipFree(device_vwlb_cs_lengths);
  hipFree(device_vwlb_cs2_lengths);
  hipFree(device_vp_cs_covered);
  hipFree(device_vp_cs_covered);
  hipFree(device_empirical_mu);
}

void bridge::copy_to_device(){
  hipMemcpy(device_dev_settings.l_vars.mu, host_dev_settings.l_vars.mu,
    K * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.l_vars.c, host_dev_settings.l_vars.c,
    n_samples * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.epsilon, host_dev_settings.epsilon,
    sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.max_n_iter, host_dev_settings.max_n_iter,
    sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.bootstrap_confidence, host_dev_settings.bootstrap_confidence,
    sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.ci_quantile, host_dev_settings.ci_quantile,
    sizeof(double), hipMemcpyHostToDevice);
}
