void bridge::count_coverage(){
  // waiting for execution of threads
  hipMemcpy(vwlb_cs_covered, device_vwlb_cs_covered,
    sizeof(int) * K * n_experiments, hipMemcpyDeviceToHost);

  hipMemcpy(vwlb_cs2_covered, device_vwlb_cs2_covered,
    sizeof(int) * K * n_experiments, hipMemcpyDeviceToHost);

  hipMemcpy(vp_cs_covered, device_vp_cs_covered,
    sizeof(int) * K * n_experiments, hipMemcpyDeviceToHost);

  construct_empirical_ci();

  for(int k = 0; k < K; k++){
    vp_cs_covered_counts[k] = 0;
    vwlb_cs_covered_counts[k] = 0;
    vwlb_cs2_covered_counts[k] = 0;
    
    for(int n = 0; n < n_experiments; n++){
      vp_cs_covered_counts[k]  += vp_cs_covered[k * n_experiments + n];
      vwlb_cs_covered_counts[k] += vwlb_cs_covered[k * n_experiments + n];
      vwlb_cs2_covered_counts[k] += vwlb_cs2_covered[k * n_experiments + n];

      if((empirical_ci[k][0] < host_empirical_mu[k*n_experiments + n])
        && (empirical_ci[k][1] > host_empirical_mu[k*n_experiments + n])
      ) empirical_ci_covered_counts[k]++;
    }
  }
}
