void bridge::count_coverage(){
  // waiting for execution of threads
  hipMemcpy(vwlb_cs_covered, device_vwlb_cs_covered,
    sizeof(int) * K * n_experiments, hipMemcpyDeviceToHost);

  hipMemcpy(vp_cs_covered, device_vp_cs_covered,
    sizeof(int) * K * n_experiments, hipMemcpyDeviceToHost);

  for(int k = 0; k < K; k++){
    for(int n = 0; n < n_experiments; n++){
      vp_cs_covered_counts[k]  += vp_cs_covered[k * n_experiments + n];
      vwlb_cs_covered_counts[k] += vwlb_cs_covered[k * n_experiments + n];
    }
  }
}
