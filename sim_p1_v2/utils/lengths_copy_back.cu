void bridge::lengths_copy_back(){
  hipMemcpy(vwlb_cs_lengths, device_vwlb_cs_lengths,
    sizeof(double) * K * n_experiments, hipMemcpyDeviceToHost);
  hipMemcpy(vwlb_cs2_lengths, device_vwlb_cs2_lengths,
    sizeof(double) * K * n_experiments, hipMemcpyDeviceToHost);
  hipMemcpy(vp_cs_lengths, device_vp_cs_lengths,
    sizeof(double) * K * n_experiments, hipMemcpyDeviceToHost);
}
