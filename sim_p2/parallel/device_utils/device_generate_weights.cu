#include "hip/hip_runtime.h"
__device__ void cavi_implementor::generate_weights(){
  hiprandState state;
  hiprand_init(thread_id, b_count + 1, 0, &state);

  double sum_weights = 0;
  for(int i = 0; i < n_samples; i++){
    sum_weights += (
      weights[i] = -log(hiprand_uniform_double(&state))
    );
  }

  for(int i = 0; i < n_samples; i++){
    weights[i] *= n_samples/sum_weights;
  }

  // by the way we can initialize estimates
}
