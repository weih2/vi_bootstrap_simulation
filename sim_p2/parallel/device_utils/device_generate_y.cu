#include "hip/hip_runtime.h"
__device__ void cavi_implementor::generate_y(){
  // set random state
  hiprandState state;
  hiprand_init(-1, thread_id, 0, &state);

  for(i = 0; i < n; i++){
    y[i] = 0;
    for(k = 0; k < p; k++){
      y[i] += true_pars.beta[k] * x[i * p + k];
    }
    y[i] += hiprand_normal_double(&state) * sqrt(true_pars.sigma_2);
  }
}
