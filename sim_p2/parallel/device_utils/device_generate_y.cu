#include "hip/hip_runtime.h"
__device__ void cavi_implementor::generate_y(){
  // set random state
  hiprandState state;
  hiprand_init(-1, thread_id, 0, &state);

  for(i = 0; i < n_samples; i++){
    y[i] = 0;
    for(k = 0; k < n_parameters; k++){
      y[i] += true_pars.beta[k] * x[i * n_parameters + k];
    }
    y[i] += hiprand_normal_double(&state) * sqrt(true_pars.sigma_2);
  }
}
