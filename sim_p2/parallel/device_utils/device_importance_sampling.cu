#include "hip/hip_runtime.h"
__device__ void cavi_implementor::importance_sampling(){
  for(int sigma_2_ind = 0; sigma_2_ind < n_sigma_2; sigma_2_ind++){
    current_pars.sigma_2 = sigma_2_lower + (sigma_2_upper - sigma_2_lower)
      * sigma_2_ind / double(n_sigma_2);
    for(int sigma_b_2_ind = 0; sigma_b_2_ind < n_sigma_b_2; sigma_b_2_ind++){
      current_pars.sigma_b_2 = sigma_b_2_lower + (sigma_b_2_upper - sigma_b_2_lower)
        * sigma_b_2_ind / double(n_sigma_b_2);
      for(int pi_ind = 0; pi_ind < n_pi; pi_ind++){
        current_pars.pi = exp10(
          log_pi_lower + (log_pi_upper - log_pi_lower) * pi_ind / double(n_pi);
        );
        // innder loop
      }
    }
  }

  // calculate posterior MAP
}
