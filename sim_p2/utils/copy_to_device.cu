void bridge::init_device(){
  hipMalloc((void**)&(device_dev_settings.g_pars), sizeof(global_pars));
  hipMalloc((void**)&(device_dev_settings.x), n * p * sizeof(double));
}

void bridge::copy_to_device(){
  hipMemcpy(device_dev_settings.g_pars, host_dev_settings.g_pars,
    sizeof(global_pars), hipMemcpyHostToDevice);
  hipMemcpy(device_dev_settings.x, host_dev_settings.x,
    sizeof(double) * n * p, hipMemcpyHostToDevice);
}

void bridge::clean_device(){
  hipFree(device_dev_settings.g_pars);
  hipFree(device_dev_settings.x);
}
